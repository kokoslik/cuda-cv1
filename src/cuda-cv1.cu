#include <iostream>
#include <cstring>
#include <fstream>
#include <algorithm>
#include <cmath>
#include <ctime>
#include <hip/hip_runtime.h>

using namespace std;
typedef unsigned char uchar;

#define NX 32
#define NY 32

static void HandleError(hipError_t err,
                        const char *file,
                        int line)
{
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err),
            file, line);
        exit(EXIT_FAILURE);
    }
}

#define HANDLE_ERROR( error ) (HandleError( error, __FILE__, __LINE__ ))

__global__ void gauss_filter(uchar* Imdev,uchar* Imnewdev,int oldwidth,int oldheight)
{
	uchar GK[]={2,4,5,4,2,
				4,9,12,9,4,
				5,12,15,12,5,
				4,9,12,9,4,
				2,4,5,4,2};
	int i=threadIdx.x+blockDim.x*blockIdx.x+2;
	int j=threadIdx.y+blockDim.y*blockIdx.y+2;
	if((i<oldheight-2)&&(j<oldwidth-2))
	{
		int res=0;
		for(int ii=0;ii<5;ii++)
			for(int jj=0;jj<5;jj++)
				res+=Imdev[(i-2+ii)*oldwidth+(j-2+jj)]*GK[ii*5+jj];
		Imnewdev[i*oldwidth+j]=res/159;
	}
}

__global__ void gradient(uchar* Imdev,uchar* Imnewdev,int oldwidth,int oldheight)
{
	int Gx[]={-1,0,1,
			  -2,0,2,
			  -1,0,1};
	int Gy[]={-1,-2,-1,
	           0, 0, 0,
			   1, 2, 1};
	int i=threadIdx.x+blockDim.x*blockIdx.x+1;
	int j=threadIdx.y+blockDim.y*blockIdx.y+1;
	if((i<oldheight-1)&&(j<oldwidth-1))
	{
		int res1=0,res2=0;
		for(int ii=0;ii<3;ii++)
			for(int jj=0;jj<3;jj++)
			{
				res1+=Imdev[(i-1+ii)*oldwidth+(j-1+jj)]*Gx[ii*3+jj];
				res2+=Imdev[(i-1+ii)*oldwidth+(j-1+jj)]*Gy[ii*3+jj];
			}
		Imnewdev[i*oldwidth+j]= __fsqrt_ru(res1*res1+res2*res2);
	}

}

float improcGPU(const char* infilename, const char* outfilename, int width,int height,int Glower,int Gupper)
{
	ifstream ifile(infilename);
	ofstream ofile(outfilename);
	float time;
	width+=4;
	height+=4;
	uchar *Im,*Imdev,*Imnewdev;
	size_t size=width*height*sizeof(uchar);
	Im=new uchar[width*height];
	memset(Im,255,size);
	hipEvent_t start,stop;
	HANDLE_ERROR( hipMalloc(&Imdev,size) );
	HANDLE_ERROR( hipMalloc(&Imnewdev,size) );
	hipEventCreate(&start);
	hipEventCreate(&stop);
	dim3 threads(NX,NY,1),blocks(height%NX==0?height/NX:height/NX+1,width%NY==0?width/NY:width/NY+1);
	for(int i=2;i<height-2;i++)
		for(int j=2;j<width-2;j++)
			ifile>>(int&)Im[i*width+j];
	ifile.close();
	hipEventRecord(start);
	HANDLE_ERROR( hipMemcpy(Imdev,Im,size,hipMemcpyHostToDevice) );
	gauss_filter<<<blocks,threads>>>(Imdev,Imnewdev,width,height);
	HANDLE_ERROR( hipGetLastError() );
	HANDLE_ERROR( hipDeviceSynchronize() );
	swap(Imdev,Imnewdev);
	gradient<<<blocks,threads>>>(Imdev,Imnewdev,width,height);
	HANDLE_ERROR( hipMemcpy(Im,Imnewdev,size,hipMemcpyDeviceToHost) );
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time,start,stop);
	for(int i=2;i<height-2;i++)
	{
			for(int j=2;j<width-2;j++)
				ofile<<(((Glower<Im[i*width+j])&&(Gupper>Im[i*width+j]))?255:0 )<<' ';
			ofile<<endl;
	}
	ofile.close();
	delete[] Im;
	HANDLE_ERROR( hipFree(Imdev) );
	HANDLE_ERROR( hipFree(Imnewdev) );
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return time*1e-3;
}

int main()
{
	cout<<"Time: "<<improcGPU("img.txt","imfilt.txt",960,512,100,200)<<endl;
return 0;
}
